
#include <hip/hip_runtime.h>
#ifdef DFTEFE_WITH_DEVICE_CUDA
#  include "DeviceAPICalls.h"
#  include <stdio.h>
namespace dftefe
{
  namespace utils
  {
#  define CUDACHECK(cmd)                              \
    do                                                \
      {                                               \
        cudaError_t e = cmd;                          \
        if (e != cudaSuccess)                         \
          {                                           \
            printf("Failed: Cuda error %s:%d '%s'\n", \
                   __FILE__,                          \
                   __LINE__,                          \
                   cudaGetErrorString(e));            \
            exit(EXIT_FAILURE);                       \
          }                                           \
      }                                               \
    while (0)

    // todo
    //  namespace {
    //    __global__ void
    //    setValueKernel(void *devPtr, int value, size_t size)
    //    {
    //
    //      const unsigned int globalThreadId =
    //        blockIdx.x * blockDim.x + threadIdx.x;
    //      for (unsigned int i = globalThreadId; i < size;
    //           i += blockDim.x * gridDim.x)
    //        {
    //          v[i] = dftefe::utils::add(dftefe::utils::mult(a, u[i]), v[i]);
    //        }
    //    }
    //  }


    void
    deviceMalloc(void **devPtr, size_t size)
    {
      CUDACHECK(cudaMalloc(devPtr, size));
    }

    void
    deviceMemset(void *devPtr, size_t count)
    {
      CUDACHECK(cudaMemset(devPtr, 0, count));
    }

    // todo
    //    void
    //    deviceSetValue(void *devPtr, int value, size_t size)
    //    {
    //
    //    }

    void
    deviceFree(void *devPtr)
    {
      CUDACHECK(cudaFree(devPtr));
    }

    void
    deviceGetDeviceCount(int *count)
    {
      CUDACHECK(cudaGetDeviceCount(count));
    }

    void
    deviceSetDevice(int count)
    {
      CUDACHECK(cudaSetDevice(count));
    }
    void
    deviceMemcpyD2H(void *dst, const void *src, size_t count)
    {
      CUDACHECK(cudaMemcpy(dst, src, count, cudaMemcpyDeviceToHost));
    }

    void
    deviceMemcpyD2D(void *dst, const void *src, size_t count)
    {
      CUDACHECK(cudaMemcpy(dst, src, count, cudaMemcpyDeviceToDevice));
    }
    void
    deviceMemcpyH2D(void *dst, const void *src, size_t count)
    {
      CUDACHECK(cudaMemcpy(dst, src, count, cudaMemcpyHostToDevice));
    }
  } // namespace utils
} // namespace dftefe
#endif
