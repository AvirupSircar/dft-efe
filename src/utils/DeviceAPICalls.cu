
#include <hip/hip_runtime.h>
#ifdef DFTEFE_WITH_DEVICE_CUDA
#  include "DeviceAPICalls.h"
#  include <stdio.h>
namespace dftefe
{
  namespace utils
  {
#  define CUDACHECK(cmd)                              \
    do                                                \
      {                                               \
        cudaError_t e = cmd;                          \
        if (e != cudaSuccess)                         \
          {                                           \
            printf("Failed: Cuda error %s:%d '%s'\n", \
                   __FILE__,                          \
                   __LINE__,                          \
                   cudaGetErrorString(e));            \
            exit(EXIT_FAILURE);                       \
          }                                           \
      }                                               \
    while (0)


    void
    deviceMalloc(void **devPtr, size_t size)
    {
      CUDACHECK(cudaMalloc(devPtr, size));
    }

    void
    deviceMemset(void *devPtr, int value, size_t count)
    {
      CUDACHECK(cudaMemset(devPtr, value, count));
    }

    void
    deviceFree(void *devPtr)
    {
      CUDACHECK(cudaFree(devPtr));
    }

    void
    deviceGetDeviceCount(int *count)
    {
      CUDACHECK(cudaGetDeviceCount(count));
    }

    void
    deviceSetDevice(int count)
    {
      CUDACHECK(cudaSetDevice(count));
    }
    void
    deviceMemcpyD2H(void *dst, const void *src, size_t count)
    {
      CUDACHECK(cudaMemcpy(dst, src, count, cudaMemcpyDeviceToHost));
    }

    void
    deviceMemcpyD2D(void *dst, const void *src, size_t count)
    {
      CUDACHECK(cudaMemcpy(dst, src, count, cudaMemcpyDeviceToDevice));
    }
    void
    deviceMemcpyH2D(void *dst, const void *src, size_t count)
    {
      CUDACHECK(cudaMemcpy(dst, src, count, cudaMemcpyHostToDevice));
    }
  } // namespace utils
} // namespace dftefe
#endif
