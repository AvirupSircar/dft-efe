
#include <hip/hip_runtime.h>
#ifdef DFTEFE_WITH_DEVICE_CUDA
#  include "DeviceAPICalls.h"
#  include <stdio.h>
#  include <vector>
#  include "DeviceDataTypeOverloads.cuh"
#  include "DeviceKernelLauncher.h"
namespace dftefe
{
  namespace utils
  {
#  define CUDACHECK(cmd)                              \
    do                                                \
      {                                               \
        cudaError_t e = cmd;                          \
        if (e != cudaSuccess)                         \
          {                                           \
            printf("Failed: Cuda error %s:%d '%s'\n", \
                   __FILE__,                          \
                   __LINE__,                          \
                   cudaGetErrorString(e));            \
            exit(EXIT_FAILURE);                       \
          }                                           \
      }                                               \
    while (0)

    namespace
    {
      template <typename ValueType>
      __global__ void
      setValueKernel(ValueType *devPtr, ValueType value, size_type size)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;
        for (unsigned int i = globalThreadId; i < size;
             i += blockDim.x * gridDim.x)
          {
            devPtr[i] = value;
          }
      }
    } // namespace

    void
    deviceGetDeviceCount(int *count)
    {
      CUDACHECK(cudaGetDeviceCount(count));
    }

    void
    deviceSetDevice(int count)
    {
      CUDACHECK(cudaSetDevice(count));
    }

    void
    deviceMalloc(void **devPtr, size_type size)
    {
      CUDACHECK(cudaMalloc(devPtr, size));
    }

    void
    deviceMemset(void *devPtr, size_type count)
    {
      CUDACHECK(cudaMemset(devPtr, 0, count));
    }

    template <typename ValueType>
    void
    deviceSetValue(ValueType *devPtr, ValueType value, size_type size)
    {
      setValueKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                       dftefe::utils::BLOCK_SIZE>>>(
        makeDataTypeDeviceCompatible(devPtr),
        makeDataTypeDeviceCompatible(value),
        size);
    }

    template void
    deviceSetValue(size_type *devPtr, size_type value, size_type size);

    template void
    deviceSetValue(int *devPtr, int value, size_type size);

    template void
    deviceSetValue(double *devPtr, double value, size_type size);

    template void
    deviceSetValue(float *devPtr, float value, size_type size);

    template void
    deviceSetValue(std::complex<float> *devPtr,
                   std::complex<float>  value,
                   size_type            size);

    template void
    deviceSetValue(std::complex<double> *devPtr,
                   std::complex<double>  value,
                   size_type             size);

    void
    deviceFree(void *devPtr)
    {
      CUDACHECK(cudaFree(devPtr));
    }

    void
    hostPinnedMalloc(void **hostPtr, size_type size)
    {
      CUDACHECK(cudaMallocHost(hostPtr, size));
    }

    void
    hostPinnedFree(void *hostPtr)
    {
      CUDACHECK(cudaFreeHost(hostPtr));
    }

    void
    deviceMemcpyD2H(void *dst, const void *src, size_type count)
    {
      CUDACHECK(cudaMemcpy(dst, src, count, cudaMemcpyDeviceToHost));
    }

    void
    deviceMemcpyD2D(void *dst, const void *src, size_type count)
    {
      CUDACHECK(cudaMemcpy(dst, src, count, cudaMemcpyDeviceToDevice));
    }
    void
    deviceMemcpyH2D(void *dst, const void *src, size_type count)
    {
      CUDACHECK(cudaMemcpy(dst, src, count, cudaMemcpyHostToDevice));
    }
  } // namespace utils
} // namespace dftefe
#endif
