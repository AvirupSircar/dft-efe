
#include <hip/hip_runtime.h>
#ifdef DFTEFE_WITH_DEVICE_CUDA
#  include <utils/DeviceKernelLauncher.h>
#  include <utils/DeviceDataTypeOverloads.cuh>
#  include <utils/DataTypeOverloads.h>
#  include <utils/MemoryTransfer.h>
#  include <utils/Exceptions.h>
#  include <linearAlgebra/BlasLapackKernels.h>
#  include <complex>
#  include <algorithm>
namespace dftefe
{
  namespace linearAlgebra
  {
    namespace blasLapack
    {
      namespace
      {
        template <typename ValueType>
        __global__ void
        axpbyDeviceKernel(const size_type  size,
                          const ValueType  alpha,
                          const ValueType *x,
                          const ValueType  beta,
                          const ValueType *y,
                          ValueType *      z)
        {
          const unsigned int globalThreadId =
            blockIdx.x * blockDim.x + threadIdx.x;
          for (unsigned int i = globalThreadId; i < size;
               i += blockDim.x * gridDim.x)
            {
              z[i] = dftefe::utils::add(dftefe::utils::mult(alpha, x[i]),
                                        dftefe::utils::mult(beta, y[i]));
            }
        }

      } // namespace



      template <typename ValueType>
      void
      Kernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::axpby(
        const size_type  size,
        const ValueType  alpha,
        const ValueType *x,
        const ValueType  beta,
        const ValueType *y,
        ValueType *      z)
      {
        axpbyDeviceKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                            dftefe::utils::BLOCK_SIZE>>>(
          size,
          dftefe::utils::makeDataTypeDeviceCompatible(alpha),
          dftefe::utils::makeDataTypeDeviceCompatible(x),
          dftefe::utils::makeDataTypeDeviceCompatible(beta),
          dftefe::utils::makeDataTypeDeviceCompatible(y),
          dftefe::utils::makeDataTypeDeviceCompatible(z));
      }


      template <typename ValueType>
      std::vector<double>
      Kernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::amaxsMultiVector(
        size_type        vecSize,
        size_type        numVec,
        ValueType const *multiVecData)
      {
        std::vector<double> amaxs(numVec, 0);

        utils::throwException(
          false,
          "amaxsMultiVector() is not implemented for dftefe::utils::MemorySpace::DEVICE .... ");
        return amaxs;
      }



      template <typename ValueType>
      std::vector<double>
      Kernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::nrms2MultiVector(
        size_type                                        vecSize,
        size_type                                        numVec,
        ValueType const *                                multiVecData,
        blasQueueType<dftefe::utils::MemorySpace::HOST> &blasQueue)
      {
        std::vector<double> nrms2(numVec, 0);
        return nrms2;
      }

      template class Kernels<double, dftefe::utils::MemorySpace::DEVICE>;
      template class Kernels<float, dftefe::utils::MemorySpace::DEVICE>;
      template class Kernels<std::complex<double>,
                             dftefe::utils::MemorySpace::DEVICE>;
      template class Kernels<std::complex<float>,
                             dftefe::utils::MemorySpace::DEVICE>;
    } // namespace blasLapack
  }   // namespace linearAlgebra
} // namespace dftefe
#endif
