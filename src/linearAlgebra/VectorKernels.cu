
#include <hip/hip_runtime.h>
#ifdef DFTEFE_WITH_DEVICE_CUDA
#  include <utils/DeviceKernelLauncher.h>
#  include <utils/DeviceDataTypeOverloads.cuh>
#  include <utils/DataTypeOverloads.h>
#  include <utils/MemoryTransfer.h>
#  include <linearAlgebra/VectorKernels.h>
#  include <linearAlgebra/DeviceBlasLapackTemplates.h>
#  include <linearAlgebra/DeviceLAContextsSingleton.h>
#  include <complex>
#  include <algorithm>
namespace dftefe
{
  namespace linearAlgebra
  {
    namespace
    {
      template <typename ValueType>
      __global__ void
      addCUDAKernel(size_type size, const ValueType *u, ValueType *v)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;
        for (unsigned int i = globalThreadId; i < size;
             i += blockDim.x * gridDim.x)
          {
            v[i] = dftefe::utils::add(v[i], u[i]);
          }
      }

      template <typename ValueType>
      __global__ void
      subCUDAKernel(size_type size, const ValueType *u, ValueType *v)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;
        for (unsigned int i = globalThreadId; i < size;
             i += blockDim.x * gridDim.x)
          {
            v[i] = dftefe::utils::sub(v[i], u[i]);
          }
      }

      template <typename ValueType>
      __global__ void
      addCUDAKernel(size_type        size,
                    const ValueType  a,
                    const ValueType *u,
                    const ValueType  b,
                    const ValueType *v,
                    ValueType *      w)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;
        for (unsigned int i = globalThreadId; i < size;
             i += blockDim.x * gridDim.x)
          {
            w[i] = dftefe::utils::add(dftefe::utils::mult(a, u[i]),
                                      dftefe::utils::mult(b, v[i]));
          }
      }

    } // namespace

    template <typename ValueType>
    void
    VectorKernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::add(
      const size_type  size,
      const ValueType *u,
      ValueType *      v)
    {
      addCUDAKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                      dftefe::utils::BLOCK_SIZE>>>(
        size,
        dftefe::utils::makeDataTypeDeviceCompatible(u),
        dftefe::utils::makeDataTypeDeviceCompatible(v));
    }

    template <typename ValueType>
    void
    VectorKernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::sub(
      const size_type  size,
      const ValueType *u,
      ValueType *      v)
    {
      subCUDAKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                      dftefe::utils::BLOCK_SIZE>>>(
        size,
        dftefe::utils::makeDataTypeDeviceCompatible(u),
        dftefe::utils::makeDataTypeDeviceCompatible(v));
    }

    template <typename ValueType>
    double
    VectorKernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::l2Norm(
      const size_type  size,
      const ValueType *u)
    {
      double l2norm = 0;
      dftefe::linearAlgebra::DeviceBlasLapack<ValueType>::nrm2(
        dftefe::linearAlgebra::DeviceLAContextsSingleton::getInstance()
          ->getDeviceBlasHandle(),
        size,
        u,
        1,
        &l2norm);
      return l2norm;
    }


    template <typename ValueType>
    double
    VectorKernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::lInfNorm(
      const size_type  size,
      const ValueType *u)
    {
      int maxIndex = 0;
      dftefe::linearAlgebra::DeviceBlasLapack<ValueType>::iamax(
        dftefe::linearAlgebra::DeviceLAContextsSingleton::getInstance()
          ->getDeviceBlasHandle(),
        size,
        u,
        1,
        &maxIndex);


      ValueType temp = 0.0;
      utils::MemoryTransfer<
        dftefe::utils::MemorySpace::HOST,
        dftefe::utils::MemorySpace::DEVICE>::copy(1, &temp, u + maxIndex - 1);

      return dftefe::utils::abs_(temp);
    }


    template <typename ValueType>
    void
    VectorKernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::add(
      size_type        size,
      ValueType        a,
      const ValueType *u,
      ValueType        b,
      const ValueType *v,
      ValueType *      w)
    {
      addCUDAKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                      dftefe::utils::BLOCK_SIZE>>>(
        size,
        dftefe::utils::makeDataTypeDeviceCompatible(a),
        dftefe::utils::makeDataTypeDeviceCompatible(u),
        dftefe::utils::makeDataTypeDeviceCompatible(b),
        dftefe::utils::makeDataTypeDeviceCompatible(v),
        dftefe::utils::makeDataTypeDeviceCompatible(w));
    }

    template <typename ValueType>
    void
    VectorKernels<ValueType, dftefe::utils::MemorySpace::HOST_PINNED>::add(
      const size_type  size,
      const ValueType *u,
      ValueType *      v)
    {
      for (size_type i = 0; i < size; ++i)
        {
          v[i] += u[i];
        }
    }

    template <typename ValueType>
    void
    VectorKernels<ValueType, dftefe::utils::MemorySpace::HOST_PINNED>::sub(
      const size_type  size,
      const ValueType *u,
      ValueType *      v)
    {
      for (size_type i = 0; i < size; ++i)
        {
          v[i] -= u[i];
        }
    }

    template <typename ValueType>
    double
    VectorKernels<ValueType, dftefe::utils::MemorySpace::HOST_PINNED>::l2Norm(
      const size_type  size,
      const ValueType *u)
    {
      double temp = 0.0;
      for (size_type i = 0; i < size; ++i)
        {
          temp += dftefe::utils::absSq(u[i]);
        }
      return std::sqrt(temp);
    }


    template <typename ValueType>
    double
    VectorKernels<ValueType, dftefe::utils::MemorySpace::HOST_PINNED>::lInfNorm(
      const size_type  size,
      const ValueType *u)
    {
      return dftefe::utils::abs_(
        *std::max_element(u, u + size, dftefe::utils::absCompare<ValueType>));
    }


    template <typename ValueType>
    void
    VectorKernels<ValueType, dftefe::utils::MemorySpace::HOST_PINNED>::add(
      size_type        size,
      ValueType        a,
      const ValueType *u,
      ValueType        b,
      const ValueType *v,
      ValueType *      w)
    {
      for (int i = 0; i < size; ++i)
        {
          w[i] = a * u[i] + b * v[i];
        }
    }

    template class VectorKernels<size_type, dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<int, dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<double, dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<float, dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<std::complex<double>,
                                 dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<std::complex<float>,
                                 dftefe::utils::MemorySpace::DEVICE>;

    template class VectorKernels<size_type,
                                 dftefe::utils::MemorySpace::HOST_PINNED>;
    template class VectorKernels<int, dftefe::utils::MemorySpace::HOST_PINNED>;
    template class VectorKernels<double,
                                 dftefe::utils::MemorySpace::HOST_PINNED>;
    template class VectorKernels<float,
                                 dftefe::utils::MemorySpace::HOST_PINNED>;
    template class VectorKernels<std::complex<double>,
                                 dftefe::utils::MemorySpace::HOST_PINNED>;
    template class VectorKernels<std::complex<float>,
                                 dftefe::utils::MemorySpace::HOST_PINNED>;
  } // namespace linearAlgebra
} // namespace dftefe
#endif
