
#include <hip/hip_runtime.h>
#ifdef DFTEFE_WITH_DEVICE_CUDA
#  include <DeviceKernelLauncher.h>
#  include "DeviceDataTypeOverloads.cuh"
#  include "VectorKernels.h"
namespace dftefe
{
  namespace linearAlgebra
  {
    namespace
    {
      template <typename ValueType>
      __global__ void
      addCUDAKernel(size_type size, const ValueType *u, ValueType *v)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;
        for (unsigned int i = globalThreadId; i < size;
             i += blockDim.x * gridDim.x)
          {
            v[i] = dftefe::utils::add(v[i], u[i]);
          }
      }

      template <typename ValueType>
      __global__ void
      subCUDAKernel(size_type size, const ValueType *u, ValueType *v)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;
        for (unsigned int i = globalThreadId; i < size;
             i += blockDim.x * gridDim.x)
          {
            v[i] = dftefe::utils::sub(v[i], u[i]);
          }
      }

      template <typename ValueType>
      __global__ void
      addCUDAKernel(size_type        size,
                    const ValueType  a,
                    const ValueType *u,
                    const ValueType  b,
                    const ValueType *v,
                    ValueType       *w)
      {
        const unsigned int globalThreadId =
          blockIdx.x * blockDim.x + threadIdx.x;
        for (unsigned int i = globalThreadId; i < size;
             i += blockDim.x * gridDim.x)
          {
            w[i] = dftefe::utils::add(dftefe::utils::mult(a, u[i]),
                                      dftefe::utils::mult(b, v[i]));
          }
      }

    } // namespace

    template <typename ValueType>
    void
    VectorKernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::add(
      const size_type  size,
      const ValueType *u,
      ValueType       *v)
    {
      addCUDAKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                      dftefe::utils::BLOCK_SIZE>>>(
        size,
        dftefe::utils::makeDataTypeDeviceCompatible(u),
        dftefe::utils::makeDataTypeDeviceCompatible(v));
    }

    template <typename ValueType>
    void
    VectorKernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::sub(
      const size_type  size,
      const ValueType *u,
      ValueType       *v)
    {
      subCUDAKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                      dftefe::utils::BLOCK_SIZE>>>(
        size,
        dftefe::utils::makeDataTypeDeviceCompatible(u),
        dftefe::utils::makeDataTypeDeviceCompatible(v));
    }


    template <typename ValueType>
    void
    VectorKernels<ValueType, dftefe::utils::MemorySpace::DEVICE>::add(
      size_type        size,
      ValueType        a,
      const ValueType *u,
      ValueType        b,
      const ValueType *v,
      ValueType       *w)
    {
      addCUDAKernel<<<size / dftefe::utils::BLOCK_SIZE + 1,
                      dftefe::utils::BLOCK_SIZE>>>(
        size,
        dftefe::utils::makeDataTypeDeviceCompatible(a),
        dftefe::utils::makeDataTypeDeviceCompatible(u),
        dftefe::utils::makeDataTypeDeviceCompatible(b),
        dftefe::utils::makeDataTypeDeviceCompatible(v),
        dftefe::utils::makeDataTypeDeviceCompatible(w));
    }

    template class VectorKernels<size_type, dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<int, dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<double, dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<float, dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<std::complex<double>,
                                 dftefe::utils::MemorySpace::DEVICE>;
    template class VectorKernels<std::complex<float>,
                                 dftefe::utils::MemorySpace::DEVICE>;
  } // namespace linearAlgebra
} // namespace dftefe
#endif
